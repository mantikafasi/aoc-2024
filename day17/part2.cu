#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
#include <device_atomic_functions.h>

#include <omp.h>
#include <stdio.h>

#include <algorithm>
#include <atomic>
#include <chrono>
#include <cmath>
#include <fstream>
#include <iostream>
#include <map>
#include <sstream>
#include <vector>

using namespace std;
using namespace std::chrono;

vector<string> readContents(string filename) {
    ifstream file(filename);

    vector<string> contents;

    string line;
    while (getline(file, line)) {
        contents.push_back(line);
    }
    return contents;
}

template <typename T>
vector<T> splitString(string& input, char delimiter) {
    istringstream stream(input);
    string token;
    vector<T> vec;

    while (getline(stream, token, delimiter)) {
        if constexpr (is_same_v<T, int>) {
            vec.push_back(stoi(token));
        }
        if constexpr (is_same_v<T, string>) {
            vec.push_back(token);
        };
    }
    return vec;
};

__device__ __forceinline__ bool processInstruction(
    const int opcode, const int64_t v, const uint64_t processed,
    uint64_t& reg_a, uint64_t& reg_b, uint64_t& reg_c, int& i) {
    switch (opcode) {
        case 0:
            reg_a >>= processed;
            break;
        case 1:
            reg_b ^= v;
            break;
        case 2:
            reg_b = processed & 7;
            break;
        case 3:
            if (reg_a) i = (v / 2) - 1;
            break;
        case 4:
            reg_b ^= reg_c;
            break;
        case 6:
            reg_b = reg_a >> processed;
            break;
        case 7:
            reg_c = reg_a >> processed;
            break;
    }
    return true;
};

__device__ __inline__ uint64_t process(uint64_t v, uint64_t reg_0, uint64_t reg_1, uint64_t reg_2) {
    if (v < 4)
        return v;
    else if (v == 4) {
        return reg_0;
    } else if (v == 5) {
        return reg_1;
    } else if (v == 6) {
        return reg_2;
    };
    return 0;
}

__global__ void searchKernel(const int* inst_array, const int inst_size,
                             const uint64_t _start_k, const uint64_t chunk_size,
                             uint64_t* result, bool* found, uint64_t* found_k) {
    uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t start_k = _start_k + tid * chunk_size;

    if (*found && start_k > *found_k) return;

    for (uint64_t k = start_k; k < start_k + chunk_size && !(*found && k > *found_k); k++) {
        uint64_t reg_0 = k;
        uint64_t reg_1 = 0;
        uint64_t reg_2 = 0;

        int j = 0;

        while (true) {
            reg_1 = reg_0 & 7;

            reg_1 ^= 7;

            reg_2 = reg_0 >> reg_1;

            reg_0 >>= 3;

            reg_1 ^= reg_2;

            reg_1 ^= 7;

            if (inst_array[j] != (reg_1 & 7)) {
                break;
            }

            if (++j == inst_size) {
                printf("Found result: %lu\n", k);
                atomicMin((unsigned long long*)result, k);
                *found = true;
                *found_k = k;
                break;
            };

            if (reg_0 == 0) break;
        }
    };
}

int main() {
    auto start = high_resolution_clock::now();

    vector<string> results = readContents("input.txt");
    uint64_t part2 = 0;

    auto e = find(results.begin(), results.end(), "");

    vector<string> registersVector(results.begin(), e);
    auto program = results[results.size() - 1];

    vector<uint64_t> original_registers;

    for (auto r : registersVector) {
        auto opcode = stoi(r.substr(r.find(':') + 1));
        original_registers.push_back(opcode);
    }
    string c = program.substr(program.find(':') + 1);

    auto instructions = splitString<int>(c, ',');
    const int inst_size = instructions.size();
    int inst_array[instructions.size()];

    copy(instructions.begin(), instructions.end(), inst_array);

    uint64_t* h_result = (uint64_t*)malloc(sizeof(uint64_t));
    bool* h_found = (bool*)malloc(sizeof(bool));
    uint64_t* h_found_k = (uint64_t*)malloc(sizeof(uint64_t));

    *h_result = UINT64_MAX;
    *h_found = false;
    *h_found_k = UINT64_MAX;

    const uint64_t max_k = 300000000000000;
    const uint64_t start_k = 0;
    const uint64_t chunk_size = 10000000ULL;

    int* d_inst_array;
    uint64_t* d_result;
    bool* d_found;
    uint64_t* d_found_k;

    hipMalloc(&d_inst_array, inst_size * sizeof(int));
    hipMalloc(&d_result, sizeof(uint64_t));
    hipMalloc(&d_found, sizeof(bool));
    hipMalloc(&d_found_k, sizeof(uint64_t));

    hipMemcpy(d_inst_array, inst_array, inst_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_result, h_result, sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(d_found, h_found, sizeof(bool), hipMemcpyHostToDevice);
    hipMemcpy(d_found_k, h_found_k, sizeof(uint64_t), hipMemcpyHostToDevice);

    const int BLOCK_SIZE = 256;
    const int GRID_SIZE = 64;

    for (uint64_t current_start_k = start_k; current_start_k < max_k; current_start_k += chunk_size * GRID_SIZE * BLOCK_SIZE) {
        if (*h_found && current_start_k > *h_found_k) break;

        auto parallel_start_time = high_resolution_clock::now();

        printf("Current start k: %lu\n", current_start_k);

        searchKernel<<<GRID_SIZE, BLOCK_SIZE>>>(d_inst_array, inst_size,
                                                current_start_k, chunk_size,
                                                d_result, d_found, d_found_k);
    
        hipDeviceSynchronize();

        hipError_t error = hipGetLastError();
        if (error != hipSuccess) {
            printf("CUDA error: %lu %s\n", current_start_k, hipGetErrorString(error));
        }

        auto parallel_end_time = high_resolution_clock::now();
        auto parallel_duration = duration_cast<milliseconds>(parallel_end_time - parallel_start_time).count();

        printf("Parallel duration: %d milliseconds for: %lu elements\n", parallel_duration, chunk_size * GRID_SIZE * BLOCK_SIZE);

        hipMemcpy(h_result, d_result, sizeof(uint64_t), hipMemcpyDeviceToHost);
        hipMemcpy(h_found, d_found, sizeof(bool), hipMemcpyDeviceToHost);
    }

    hipFree(d_inst_array);
    hipFree(d_result);
    hipFree(d_found);

    cout << "\n";
    part2 = *h_result;

    cout << "Part 2: " << part2 << "\n";

    auto end = high_resolution_clock::now();
    auto duration = duration_cast<seconds>(end - start).count();

    cout << "Time taken: " << duration << " seconds" << endl;
}